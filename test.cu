#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define checkCudaErrors(call)                                 \
  do {                                                        \
    hipError_t err = call;                                   \
    if (err != hipSuccess) {                                 \
      printf("CUDA error at %s %d: %s\n", __FILE__, __LINE__, \
             hipGetErrorString(err));                        \
      exit(EXIT_FAILURE);                                     \
    }                                                         \
  } while (0)

__global__ void gpuVecAdd(float *A, float *B, float *C) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    C[tid] = A[tid] + B[tid];
}

void init(float *V, int N) {
    for (int i = 0; i < N; i++) {
        V[i] = i + 1.0;
    }
}

void calc_cpu(float* a, float* b, float* c, int N) {
    for (int i = 0; i < N; i++) {
        c[i] = a[i] + b[i];
    }
}

void validation(float* cpu, float* gpu, int N) {
    for (int i = 0; i < N; i++) {
        if (cpu[i] - gpu[i] > 1e-05) {
            printf("failed! index : %d\n", i);
            printf("cpu : %f, gpu : %f\n", cpu[i], gpu[i]);
            break;
        }
    }
}

int main() {
    int GPU_N = 0;
    checkCudaErrors(hipGetDeviceCount(&GPU_N));
    printf("CUDA-capable device count : %d\n", GPU_N);

    int N = 1048576 * 32;
    float *A = nullptr;
    float *B = nullptr;
    float **C = nullptr;
    float **cpu = nullptr;

    A = (float*)malloc(sizeof(float) * N);
    B = (float*)malloc(sizeof(float) * N);
    C = (float**)malloc(sizeof(float*) * GPU_N);
    cpu = (float**)malloc(sizeof(float*) * GPU_N);
    init(A, N);
    init(B, N);

    for (int i = 0; i < GPU_N; i++) {
        C[i] = (float*)malloc(sizeof(float) * N);
        cpu[i] = (float*)malloc(sizeof(float) * N);
        calc_cpu(A, B, cpu[i], N);
    }
    //memory objects alloc
    printf("Starting a vector addition in gpu\n");
    float *d_A[GPU_N], *d_B[GPU_N], *d_C[GPU_N];
    for (int i = 0; i < GPU_N; i++) {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipMalloc(&d_A[i], sizeof(float) * N));
        checkCudaErrors(hipMalloc(&d_B[i], sizeof(float) * N));
        checkCudaErrors(hipMalloc(&d_C[i], sizeof(float) * N));
    }

    for (int i = 0; i < GPU_N; i++) {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipMemcpy(d_A[i], A, sizeof(float) * N, hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_B[i], B, sizeof(float) * N, hipMemcpyHostToDevice));
    }

    dim3 dimBlock(32,1);
    dim3 dimGrid(N / 32, 1);

    for (int i = 0; i < GPU_N; i++) {
        checkCudaErrors(hipSetDevice(i));
        gpuVecAdd<<< dimGrid, dimBlock, 0 >>> (d_A[i], d_B[i], d_C[i]);
        checkCudaErrors(hipMemcpy(C[i], d_C[i], sizeof(float) * N, hipMemcpyDeviceToHost));
    }

    printf("Start validation\n");
    for (int i = 0; i < GPU_N; i++) {
        checkCudaErrors(hipSetDevice(i));
        validation(cpu[i], C[i], N);
    }
    printf("End validation!\n");

    for (int i = 0; i < GPU_N; i++) {
        checkCudaErrors(hipSetDevice(i));
        checkCudaErrors(hipFree(d_A[i]));
        checkCudaErrors(hipFree(d_B[i]));
        checkCudaErrors(hipFree(d_C[i]));
    }

    free(A);
    free(B);
    for (int i = 0; i < GPU_N; i++) {
        free(C[i]);
        free(cpu[i]);
    }
    return 0;
}
